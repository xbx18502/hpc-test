#include <cstdlib> // For rand() and srand()
#include <ctime>   // For time()
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrixMultiplyKernel(double *A, double *B, double *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        double sum = 0.0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void multiplyMatricesCUDA(const double* A, const double* B, double* C, int N) {
    double *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc((void **)&d_A, N*N*sizeof(double));
    hipMalloc((void **)&d_B, N*N*sizeof(double));
    hipMalloc((void **)&d_C, N*N*sizeof(double));

    // Copy data from host to device
    hipMemcpy(d_A, A, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*sizeof(double), hipMemcpyHostToDevice);

    // Setup grid and block dimensions
    dim3 dimBlock(16, 16); // Block size of 16x16
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);

    // Launch the kernel
    matrixMultiplyKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(C, d_C, N*N*sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    const int N = 1024; // Example size
    double *A, *B, *C;

    // Allocate matrices
    A = new double[N*N];
    B = new double[N*N];
    C = new double[N*N];

    // Seed random number generator
    srand(time(NULL));

    // Initialize matrices A and B with random double values
    for (int i = 0; i < N * N; ++i) {
        A[i] = rand() % 100;  // Random integers from 0 to 99
        B[i] = rand() % 100;  // Random integers from 0 to 99
    }

    // Call the CUDA matrix multiplication function
    multiplyMatricesCUDA(A, B, C, N);

    // Clean up resources
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
