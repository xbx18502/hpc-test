#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

int main(int argc, char **argv) {
    int n  = strtol(argv[1], NULL, 10);  // size of array
    printf("size = %d\n",n);
    hipblasHandle_t handle;
    float *d_A, *d_B, *d_C;
    float *A, *B, *C; // array in host memory
    float alpha = 1.0f, beta = 0.0f;
    size_t size = n * n * sizeof(float);

    A = new float[n*n];
    B = new float[n*n];
    C = new float[n*n]; // Allocate host memory to store result

    // Initialize A , B
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            A[i*n+j] = float(i + j);
            B[i*n+j] = float(i - j);
        }
    }

    // Allocate A ,B ,C on GPU
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // copy A and B to GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    hipblasCreate(&handle);

    clock_t start = clock();
    //  C = alpha * A * B + beta * C
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);
    clock_t end = clock();
    // copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("size = %d, time :%f second\n",n,time_spent);
    
    // print part of matrix
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            printf("%f ",C[i * n + j]) ;
        }
        std::cout << "\n";
    }

    // free resources
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}

/*
module load cuda/12.0.0
nvcc -o matmul_cuBLAS.elf  matmul_cuBLAS.cu -lcublas

*/
